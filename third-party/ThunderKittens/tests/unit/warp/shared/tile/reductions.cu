#include "hip/hip_runtime.h"
#include "reductions.cuh"

#ifdef TEST_WARP_SHARED_TILE_REDUCTIONS

template<typename T>
struct normalize_row {
    using dtype = T;
    template<int H, int W, int NW> using valid = std::bool_constant<NW == 1 && W*H<=64 && sizeof(dtype) != 1>; // this is warp-level
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "shared_norm_row_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "shared_norm_row_gmem=half" :
                                                                                         "shared_norm_row_gmem=float";
    template<int H, int W, int NW, gl_t GL> __host__ static void host_func(const std::vector<float> &i_ref_f, std::vector<float> &o_ref_f) {
        if constexpr(std::is_same_v<dtype, kittens::bf16>) {
            std::vector<kittens::bf16> i_ref(i_ref_f.size());
            std::vector<kittens::bf16> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = __float2bfloat16(i_ref_f[i]);
            for(int i = 0; i < H*16; i++) {
                kittens::bf16 row_sum = 0;
                for(int j = 0; j < W*16; j++) {
                    o_ref[i*W*16+j]  = i_ref[i*W*16+j];
                    row_sum         += i_ref[i*W*16+j];
                }
                row_sum = __habs(row_sum)+__float2bfloat16(1.f);
                for(int j = 0; j < W*16; j++) o_ref[i*W*16+j] /= row_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = __bfloat162float(o_ref[i]);
        }
        else if constexpr(std::is_same_v<dtype, float>) {
            std::vector<float> i_ref(i_ref_f.size());
            std::vector<float> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = i_ref_f[i];
            for(int i = 0; i < H*16; i++) {
                float row_sum = 0;
                for(int j = 0; j < W*16; j++) {
                    o_ref[i*W*16+j]  = i_ref[i*W*16+j];
                    row_sum         += i_ref[i*W*16+j];
                }
                row_sum = abs(row_sum)+1.f;
                for(int j = 0; j < W*16; j++) o_ref[i*W*16+j] /= row_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = o_ref[i];
        }
        else {
            std::vector<kittens::half> i_ref(i_ref_f.size());
            std::vector<kittens::half> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = __float2half(i_ref_f[i]);
            for(int i = 0; i < H*16; i++) {
                kittens::half row_sum = 0;
                for(int j = 0; j < W*16; j++) {
                    o_ref[i*W*16+j]  = i_ref[i*W*16+j];
                    row_sum         += i_ref[i*W*16+j];
                }
                row_sum = __habs(row_sum)+__float2half(1.f);
                for(int j = 0; j < W*16; j++) o_ref[i*W*16+j] /= row_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = __half2float(o_ref[i]);
        }
    }
    template<int H, int W, int NW, gl_t GL> __device__ static void device_func(const GL &input, const GL &output) {
        extern __shared__ kittens::alignment_dummy __shm[];
        kittens::shared_allocator al((int*)&__shm[0]); 
        kittens::st<dtype, 16*H, 16*W> &shared_tile = al.allocate<kittens::st<dtype, 16*H, 16*W>>();
        __shared__ kittens::col_vec<typeof(shared_tile)> accum;
        kittens::load(shared_tile, input, {});
        kittens::row_sum(accum, shared_tile);
        kittens::abs(accum, accum);
        kittens::add(accum, accum, kittens::base_types::constants<dtype>::one());
        kittens::div_row(shared_tile, shared_tile, accum);
        kittens::store(output, shared_tile, {});
    }
};
template<typename T>
struct normalize_col {
    using dtype = T;
    template<int H, int W, int NW> using valid = std::bool_constant<NW == 1 && W*H<=64 && sizeof(dtype) != 1>; // this is warp-level
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "shared_norm_col_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "shared_norm_col_gmem=half" :
                                                                                         "shared_norm_col_gmem=float";
    template<int H, int W, int NW, gl_t GL> __host__ static void host_func(const std::vector<float> &i_ref_f, std::vector<float> &o_ref_f) {
        if constexpr(std::is_same_v<dtype, kittens::bf16>) {
            std::vector<kittens::bf16> i_ref(i_ref_f.size());
            std::vector<kittens::bf16> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = __float2bfloat16(i_ref_f[i]);
            for(int i = 0; i < W*16; i++) {
                kittens::bf16 col_sum = 0;
                for(int j = 0; j < H*16; j++) {
                    o_ref[i+j*W*16]  = i_ref[i+j*W*16];
                    col_sum         += i_ref[i+j*W*16];
                }
                col_sum = __habs(col_sum)+__float2bfloat16(1.f);
                for(int j = 0; j < H*16; j++) o_ref[i+j*W*16] /= col_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = __bfloat162float(o_ref[i]);
        }
        else if constexpr(std::is_same_v<dtype, float>) {
            std::vector<float> i_ref(i_ref_f.size());
            std::vector<float> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = i_ref_f[i];
            for(int i = 0; i < W*16; i++) {
                float col_sum = 0;
                for(int j = 0; j < H*16; j++) {
                    o_ref[i+j*W*16]  = i_ref[i+j*W*16];
                    col_sum         += i_ref[i+j*W*16];
                }
                col_sum = abs(col_sum)+1.f;
                for(int j = 0; j < H*16; j++) o_ref[i+j*W*16] /= col_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = o_ref[i];
        }
        else {
            std::vector<kittens::half> i_ref(i_ref_f.size());
            std::vector<kittens::half> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = __float2half(i_ref_f[i]);
            for(int i = 0; i < W*16; i++) {
                kittens::half col_sum = 0;
                for(int j = 0; j < H*16; j++) {
                    o_ref[i+j*W*16]  = i_ref[i+j*W*16];
                    col_sum         += i_ref[i+j*W*16];
                }
                col_sum = __habs(col_sum)+__float2half(1.f);
                for(int j = 0; j < H*16; j++) o_ref[i+j*W*16] /= col_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = __half2float(o_ref[i]);
        }
    }
    template<int H, int W, int NW, gl_t GL> __device__ static void device_func(const GL &input, const GL &output) {
        extern __shared__ kittens::alignment_dummy __shm[];
        kittens::shared_allocator al((int*)&__shm[0]); 
        kittens::st<dtype, 16*H, 16*W> &shared_tile = al.allocate<kittens::st<dtype, 16*H, 16*W>>();
        __shared__ kittens::row_vec<typeof(shared_tile)> accum;
        kittens::load(shared_tile, input, {});
        kittens::col_sum(accum, shared_tile);
        kittens::abs(accum, accum);
        kittens::add(accum, accum, kittens::base_types::constants<dtype>::one());
        kittens::div_col(shared_tile, shared_tile, accum);
        kittens::store(output, shared_tile, {});
    }
};
template<typename T>
struct broadcast_row {
    using dtype = T;
    template<int H, int W, int NW> using valid = std::bool_constant<NW == 1 && W*H<=64 && sizeof(dtype) != 1>; // this is warp-level
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "shared_broadcast_row_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "shared_broadcast_row_gmem=half" :
                                                                                         "shared_broadcast_row_gmem=float";
    template<int H, int W, int NW, gl_t GL> __host__ static void host_func(const std::vector<float> &i_ref_f, std::vector<float> &o_ref_f) {
        if constexpr(std::is_same_v<dtype, kittens::bf16>) {
            std::vector<kittens::bf16> i_ref(i_ref_f.size());
            std::vector<kittens::bf16> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = __float2bfloat16(i_ref_f[i]/8.f);
            for(int i = 0; i < H*16; i++) {
                kittens::bf16 row_sum = 0;
                for(int j = 0; j < W*16; j++) {
                    o_ref[i*W*16+j]  = i_ref[i*W*16+j];
                    row_sum         += i_ref[i*W*16+j];
                }
                for(int j = 0; j < W*16; j++) o_ref[i*W*16+j] = row_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = __bfloat162float(o_ref[i]);
        }
        else if constexpr(std::is_same_v<dtype, float>) {
            std::vector<float> i_ref(i_ref_f.size());
            std::vector<float> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = i_ref_f[i]/8.f;
            for(int i = 0; i < H*16; i++) {
                float row_sum = 0;
                for(int j = 0; j < W*16; j++) {
                    o_ref[i*W*16+j]  = i_ref[i*W*16+j];
                    row_sum         += i_ref[i*W*16+j];
                }
                for(int j = 0; j < W*16; j++) o_ref[i*W*16+j] = row_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = o_ref[i];
        }
        else {
            std::vector<kittens::half> i_ref(i_ref_f.size());
            std::vector<kittens::half> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = __float2half(i_ref_f[i]/8.f);
            for(int i = 0; i < H*16; i++) {
                kittens::half row_sum = 0;
                for(int j = 0; j < W*16; j++) {
                    o_ref[i*W*16+j]  = i_ref[i*W*16+j];
                    row_sum         += i_ref[i*W*16+j];
                }
                for(int j = 0; j < W*16; j++) o_ref[i*W*16+j] = row_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = __half2float(o_ref[i]);
        }
    }
    template<int H, int W, int NW, gl_t GL> __device__ static void device_func(const GL &input, const GL &output) {
        extern __shared__ kittens::alignment_dummy __shm[];
        kittens::shared_allocator al((int*)&__shm[0]); 
        kittens::st<dtype, 16*H, 16*W> &shared_tile = al.allocate<kittens::st<dtype, 16*H, 16*W>>();
        __shared__ kittens::col_vec<typeof(shared_tile)> accum;
        kittens::load(shared_tile, input, {});
        kittens::mul(shared_tile, shared_tile, kittens::base_types::convertor<dtype, float>::convert(0.125f));
        kittens::row_sum(accum, shared_tile);
        kittens::broadcast_row(shared_tile, accum);
        kittens::store(output, shared_tile, {});
    }
};

template<typename T>
struct broadcast_col {
    using dtype = T;
    template<int H, int W, int NW> using valid = std::bool_constant<NW == 1 && W*H<=64 && sizeof(dtype) != 1>; // this is warp-level
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "shared_broadcast_col_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "shared_broadcast_col_gmem=half" :
                                                                                         "shared_broadcast_col_gmem=float";
    template<int H, int W, int NW, gl_t GL> __host__ static void host_func(const std::vector<float> &i_ref_f, std::vector<float> &o_ref_f) {
        if constexpr(std::is_same_v<dtype, kittens::bf16>) {
            std::vector<kittens::bf16> i_ref(i_ref_f.size());
            std::vector<kittens::bf16> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = __float2bfloat16(i_ref_f[i]/8.f);
            for(int i = 0; i < W*16; i++) {
                kittens::bf16 col_sum = 0;
                for(int j = 0; j < H*16; j++) {
                    o_ref[i+j*W*16]  = i_ref[i+j*W*16];
                    col_sum         += i_ref[i+j*W*16];
                }
                for(int j = 0; j < H*16; j++) o_ref[i+j*W*16] = col_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = __bfloat162float(o_ref[i]);
        }
        else if constexpr(std::is_same_v<dtype, float>) {
            std::vector<float> i_ref(i_ref_f.size());
            std::vector<float> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = i_ref_f[i]/8.f;
            for(int i = 0; i < W*16; i++) {
                float col_sum = 0;
                for(int j = 0; j < H*16; j++) {
                    o_ref[i+j*W*16]  = i_ref[i+j*W*16];
                    col_sum         += i_ref[i+j*W*16];
                }
                for(int j = 0; j < H*16; j++) o_ref[i+j*W*16] = col_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = o_ref[i];
        }
        else {
            std::vector<kittens::half> i_ref(i_ref_f.size());
            std::vector<kittens::half> o_ref(o_ref_f.size());
            for(int i = 0; i < i_ref.size(); i++) i_ref[i] = __float2half(i_ref_f[i]/8.f);
            for(int i = 0; i < W*16; i++) {
                kittens::half col_sum = 0;
                for(int j = 0; j < H*16; j++) {
                    o_ref[i+j*W*16]  = i_ref[i+j*W*16];
                    col_sum         += i_ref[i+j*W*16];
                }
                for(int j = 0; j < H*16; j++) o_ref[i+j*W*16] = col_sum;
            }
            for(int i = 0; i < o_ref.size(); i++) o_ref_f[i] = __half2float(o_ref[i]);
        }
    }
    template<int H, int W, int NW, gl_t GL> __device__ static void device_func(const GL &input, const GL &output) {
        extern __shared__ kittens::alignment_dummy __shm[];
        kittens::shared_allocator al((int*)&__shm[0]); 
        kittens::st<dtype, 16*H, 16*W> &shared_tile = al.allocate<kittens::st<dtype, 16*H, 16*W>>();
        __shared__ kittens::row_vec<typeof(shared_tile)> accum;
        kittens::load(shared_tile, input, {});
        kittens::mul(shared_tile, shared_tile, kittens::base_types::convertor<dtype, float>::convert(0.125f));
        kittens::col_sum(accum, shared_tile);
        kittens::broadcast_col(shared_tile, accum);
        kittens::store(output, shared_tile, {});
    }
};

void warp::shared::tile::reductions::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/warp/shared/tile/reductions tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;
    sweep_gmem_type_2d_warp<normalize_row, SIZE, SIZE>::run(results);
    sweep_gmem_type_2d_warp<normalize_col, SIZE, SIZE>::run(results);
    sweep_gmem_type_2d_warp<broadcast_row, SIZE, SIZE>::run(results);
    sweep_gmem_type_2d_warp<broadcast_col, SIZE, SIZE>::run(results);
}

#endif