#include "hip/hip_runtime.h"
#include "maps.cuh"

#ifdef TEST_WARP_SHARED_TILE_MAPS

template<typename T>
struct test_exp {
    using dtype = T;
    template<int H, int W, int NW> using valid = std::bool_constant<( NW == 1 && W*H<=64 && sizeof(dtype) != 1 )>; // this is warp-level
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "shared_exp_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "shared_exp_gmem=half" :
                                                                                         "shared_exp_gmem=float";
    template<int H, int W, int NW, gl_t GL> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        for(int i = 0; i < i_ref.size(); i++) o_ref[i] = __bfloat162float(__float2bfloat16(::expf(i_ref[i]))); // overwrite the whole thing
    }
    template<int H, int W, int NW, gl_t GL> __device__ static void device_func(const GL &input, const GL &output) {
        extern __shared__ kittens::alignment_dummy __shm[];
        kittens::shared_allocator al((int*)&__shm[0]); 
        kittens::st<dtype, 16*H, 16*W> &shared_tile = al.allocate<kittens::st<dtype, 16*H, 16*W>>();
        kittens::load(shared_tile, input, {});
        kittens::exp(shared_tile, shared_tile);
        kittens::store(output, shared_tile, {});
    }
};

void warp::shared::tile::maps::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/warp/shared/tile/maps tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;
    sweep_gmem_type_2d_warp<test_exp, SIZE, SIZE>::run(results);
}

#endif